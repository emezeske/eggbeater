#include "hip/hip_runtime.h"
#include <stdio.h>

const unsigned
    BLOCK_DATA_SIZE = 16,
    BLOCK_SIZE      = 80,

    SESSION_KEY_BYTES = 44,
    SESSION_KEY_WORDS = SESSION_KEY_BYTES / sizeof(unsigned);

__device__ unsigned rotl(const unsigned value, const unsigned shift)
{
    return (value << shift) | (value >> (32 - shift));
}

__device__ unsigned reorder_bytes(const unsigned char* s, const unsigned i, const unsigned end)
{
    unsigned char a = 0, b = 0, c = 0, d = 0;
    // Efficiently handle the possibility that the string buffer does not have four
    // bytes left to reorder.  Any missing bytes default to zero.
    switch (end - i)
    {
        default:
        case 4:
            d = s[i+3];
        case 3:
            c = s[i+2];
        case 2:
            b = s[i+1];
        case 1:
            a = s[i];
        case 0:
            break;
    }
    return (a<<24) | (b<<16) | (c<<8) | d;
}

struct sha1_vector
{
    static const unsigned
        NOTHING_UP_MY_SLEEVE_A = 0x67452301,
        NOTHING_UP_MY_SLEEVE_B = 0xEFCDAB89,
        NOTHING_UP_MY_SLEEVE_C = 0x98BADCFE,
        NOTHING_UP_MY_SLEEVE_D = 0x10325476,
        NOTHING_UP_MY_SLEEVE_E = 0xC3D2E1F0,

        ROUND_1_MAGIC = 0x5A827999,
        ROUND_2_MAGIC = 0x6ED9EBA1,
        ROUND_3_MAGIC = 0x8F1BBCDC,
        ROUND_4_MAGIC = 0xCA62C1D6;

    __device__ sha1_vector() :
        a(NOTHING_UP_MY_SLEEVE_A),
        b(NOTHING_UP_MY_SLEEVE_B),
        c(NOTHING_UP_MY_SLEEVE_C),
        d(NOTHING_UP_MY_SLEEVE_D),
        e(NOTHING_UP_MY_SLEEVE_E)
    {}

    __device__ sha1_vector(const sha1_vector& rhs) :
        a(rhs.a),
        b(rhs.b),
        c(rhs.c),
        d(rhs.d),
        e(rhs.e)
    {}

    __device__ unsigned round_1_f() const { return (b&c)|((~b)&d); }
    __device__ unsigned round_2_f() const { return b^c^d; }
    __device__ unsigned round_3_f() const { return (b&c)|(b&d)|(c&d); }
    __device__ unsigned round_4_f() const { return b^c^d; }

    __device__ void compute(const unsigned* message, const unsigned start, const unsigned end)
    {
        for (unsigned i = start; i < min(20, end); ++i)
        {
            compress(round_1_f(), ROUND_1_MAGIC, message[i]);
        }

        for (unsigned i = max(20, start); i < min(40, end); ++i)
        {
            compress(round_2_f(), ROUND_2_MAGIC, message[i]);
        }

        for (unsigned i = max(40, start); i < min(60, end); ++i)
        {
            compress(round_3_f(), ROUND_3_MAGIC, message[i]);
        }

        for (unsigned i = max(60, start); i < min(80, end); ++i)
        {
            compress(round_4_f(), ROUND_4_MAGIC, message[i]);
        }
    }

    __device__ void compress(const unsigned F, const unsigned k, const unsigned w)
    {
        unsigned temp = rotl(a, 5) + F + e + k + w;
        e = d;
        d = c;
        c = rotl(b, 30);
        b = a;
        a = temp;
    }

    __device__ void finalize()
    {
        a += NOTHING_UP_MY_SLEEVE_A;
        b += NOTHING_UP_MY_SLEEVE_B;
        c += NOTHING_UP_MY_SLEEVE_C;
        d += NOTHING_UP_MY_SLEEVE_D;
        e += NOTHING_UP_MY_SLEEVE_E;
    }

    __device__ bool compare(const unsigned* meebo_digest) const
    {
        // Note that the meebo digest only has 14 bytes of significant
        // values (6 bytes less than a full SHA-1 hash).
        return
            a == meebo_digest[0] &&
            b == meebo_digest[1] &&
            c == meebo_digest[2] &&
            (d & 0xffff0000) == (meebo_digest[3] & 0xffff0000);
    }

    __device__ void print() const
    {
        printf("%08x%08x%08x%08x%08x\n", a, b, c, d, e);
    }

    unsigned a, b, c, d, e;
};

struct sha1_partial_state
{
    // The initial part of the SHA-1 hash calculation, for the session key,
    // will always have the same partial result.  This function calculates it
    // and stores it so that it can be cached and reused.
    __device__ sha1_partial_state(const unsigned char *session_key)
    {
        for (unsigned i = 0; i < SESSION_KEY_WORDS; ++i)
        {
            consumed[i] =
                reorder_bytes(session_key, i * sizeof(unsigned), SESSION_KEY_BYTES);
        }

        v.compute(consumed, 0, SESSION_KEY_WORDS);
    }

    sha1_vector v;

    unsigned consumed[SESSION_KEY_WORDS];
};

// NOTE: This only supports base36 numbers with 9 digits or less.
//       Also, 's' must have enough room for the 9 digits.
__device__ unsigned ultoa36(unsigned long value, unsigned char* s)
{
    const unsigned char* base36 = "abcdefghijklmnopqrstuvwxyz0123456789";
    unsigned long base = 101559956668416; // 36 ^ 9
    bool leading_zeros = true;
    unsigned char* next = s;

    if (!value)
    {
        *next++ = base36[0];
    }
    else
    {
        while (base)
        {
            unsigned long r = value / base;
            if (r)
            {
                leading_zeros = false;
                *next++ = base36[r];
                value %= base;
            }
            else if (!leading_zeros)
            {
                *next++ = base36[r];
            }
            base /= 36;
        }
    }

    return next - s;
}

__device__ bool try_secret(
    const sha1_partial_state* precalculated_state,
    const unsigned long secret_number,
    const unsigned* meebo_digest)
{
    unsigned char append[10];
    const unsigned secret_len = ultoa36(secret_number, append);
    const unsigned append_len = secret_len + 1;
    // Add the terminating '1' bit.
    append[secret_len] = 0x80;

    if (sizeof(append) < append_len)
    {
        printf("WARNING: static append buffer is too small.\n");
        return false;
    }

    unsigned block[BLOCK_SIZE];
    unsigned* next = block;

#pragma unroll
    for (unsigned i = 0; i < SESSION_KEY_WORDS; ++i)
    {
        *next++ = precalculated_state->consumed[i];
    }

    for (unsigned i = 0; i < append_len; i += sizeof(unsigned))
    {
        *next++ = reorder_bytes(append, i, append_len);
    }

    while (next != &block[BLOCK_DATA_SIZE - 1])
    {
        *next++ = 0;
    }

    // Append the length (in bits) to the end of the message.  In reality,
    // this is a 64 bit integer, but we'll never need the upper 32 bits.
    block[BLOCK_DATA_SIZE - 1] = 32 * SESSION_KEY_WORDS + 8 * secret_len;

    // Extend the initial block contents into a full block.
#pragma unroll
    for (unsigned i = BLOCK_DATA_SIZE; i < BLOCK_SIZE; ++i)
    {
        block[i] = block[i-3] ^ block[i-8] ^ block[i-14] ^ block[i-16];
        block[i] = rotl(block[i], 1);
    }

    sha1_vector v(precalculated_state->v);
    v.compute(block, SESSION_KEY_WORDS, 80); 
    v.finalize();
    if (v.compare(meebo_digest))
    {
        append[secret_len] = 0;
        printf("Key found: %s: ", append);
        v.print();
        return true;
    }

    return false;
}

__global__ void precalculate(
    const unsigned char* session_key,
    sha1_partial_state* precalculated_state)
{
    *precalculated_state = sha1_partial_state(session_key);
}

__global__ void crack(
    const sha1_partial_state* precalculated_state,
    const unsigned* meebo_digest,
    const unsigned long* secret_number_base,
    unsigned* key_found)
{
    const unsigned long secret_number =
        *secret_number_base + blockIdx.x * blockDim.x + threadIdx.x;

    if (try_secret(precalculated_state, secret_number, meebo_digest))
    {
        *key_found = 1;
    }
}
